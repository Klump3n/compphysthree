#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>

/*
 * Dieses Beispiel demonstriert die Addition zweier Arrays.
 * addArrayGPU soll die Arbeit ueber CUDA Threads auf der GPU verteilen.
 * addArrayHost iteriert sequentiell durch die Vektorelemente auf dem Host.
 */

// Macro zur Fehlerauswertung
#define CHECK(call)                                                            \
{                                                                              \
    const hipError_t error = call;                                            \
    if (error != hipSuccess)                                                  \
    {                                                                          \
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                 \
        fprintf(stderr, "code: %d, reason: %s\n", error,                       \
                hipGetErrorString(error));                                    \
        exit(1);                                                               \
    }                                                                          \
}

double seconds()
{
    struct timeval tp;
    struct timezone tzp;
    int i = gettimeofday(&tp, &tzp);
    return ((double)tp.tv_sec + (double)tp.tv_usec * 1.e-6);
}

void checkResult(float *hostRef, float *gpuRef, const int N)
{
    double epsilon = 1.0E-8;
    bool match = 1;

    for (int i = 0; i < N; i++)
    {
        if (abs(hostRef[i] - gpuRef[i]) > epsilon)
        {
            match = 0;
            printf("Arrays stimmen nicht ueberein!\n");
            printf("host %5.2f gpu %5.2f an der Stelle %d\n", hostRef[i],
                   gpuRef[i], i);
            break;
        }
    }

    if (match) printf("Arrays stimmen ueberein.\n\n");

    return;
}

void initialData(float *ip, int size)
{
    // erzeuge zufaellige Eintraege
    time_t t;
    srand((unsigned) time(&t));

    for (int i = 0; i < size; i++)
    {
        ip[i] = (float)(rand() & 0xFF) / 10.0f;
    }

    return;
}

void addArrayHost(float *A, float *B, float *C, const int N)
{
    for (int idx = 0; idx < N; idx++)
        C[idx] = A[idx] + B[idx];
}

__global__
void addArrayGPU_new(float *A, float *B, float *C) {

  int blockOffset = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x * blockDim.y;
  int threadId = threadIdx.x + threadIdx.y * blockDim.x;
  int idx = blockOffset + threadId;

  C[idx] = A[idx] + B[idx];
}

void testGridParamter(const int nElem, FILE* results_file, int blockIdx, int blockIdy, int threadIdx, int threadIdy) {

  // Host-Speicher allozieren mit malloc
  size_t nBytes = nElem * sizeof(float);

  float latenzHDArr, latenzDHArr,
    bandbreiteHDArr, bandbreiteDHArr,
    durchsatzDArr/* , durchsatzHArr */;

  float *h_A, *h_B, *hostRef, *gpuRef;
  h_A     = (float *)malloc(nBytes);
  h_B     = (float *)malloc(nBytes);
  hostRef = (float *)malloc(nBytes);
  gpuRef  = (float *)malloc(nBytes);

  // initialisiere Arrays auf dem Host
  initialData(h_A, nElem);
  initialData(h_B, nElem);

  /* memset(hostRef, 0, nBytes); */
  memset(gpuRef,  0, nBytes);

  // Device-Speicher allozieren mit hipMalloc
  float *d_A, *d_B, *d_C;
  CHECK(hipMalloc((float**)&d_A, nBytes));
  CHECK(hipMalloc((float**)&d_B, nBytes));
  CHECK(hipMalloc((float**)&d_C, nBytes));

  // Starte Zeitmessung Latenz Host->Device
  double t_HDstart = seconds();

  // kopieren Host -> Device mit hipMemcpy
  CHECK(hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice));
  CHECK(hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice));
  CHECK(hipMemcpy(d_C, gpuRef, nBytes, hipMemcpyHostToDevice));

  // Beende Zeitmessung Latenz Host->Device
  double t_HDend = seconds();

  latenzHDArr= (t_HDend-t_HDstart)*1.e+3;

  // Berechne Bandbreite aus Latenz und Groesse der Arrays
  bandbreiteHDArr = 3*nElem*sizeof(float)/latenzHDArr*1.e-9*1.e+3; // GByte/s


  //Starte Zeitmessung Durchsatz Device
  double t_DurchD_start = seconds();

  dim3 block(blockIdx, blockIdy);
  dim3 grid(threadIdx, threadIdy);
  addArrayGPU_new <<<block, grid>>> (d_A, d_B, d_C);

  hipDeviceSynchronize();

  //Beende Zeitmessung Durchsatz Device
  double t_DurchD_end = seconds();
  double t_DurchD = t_DurchD_end - t_DurchD_start;

  durchsatzDArr = 3*nElem*1.e-9 /t_DurchD; //Faktor wegen 3 flop

  // Starte Zeitmessung Latenz Device->Host
  double t_DHstart = seconds();

  // kopieren Device -> Host mit hipMemcpy
  CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));

  // Beende Zeitmessung Latenz Device->Host
  double t_DHend = seconds();

  latenzDHArr = (t_DHend-t_DHstart)*1.e+3;

  // Berechne Bandbreite aus Latenz und Groesse der Arrays
  bandbreiteDHArr = 3*nElem*sizeof(float)/latenzDHArr*1.e-9*1.e+3; // GByte/s
  //printf(" Groesse float %lu \n", sizeof(float));

  /* //Starte Zeitmessung Durchsatz Host */
  /* double t_DurchH_start = seconds(); */

  /* // Addition auf dem Host */
  /* addArrayHost(h_A, h_B, hostRef, nElem); */

  /* //Beende Zeitmessung Durchsatz Host */
  /* double t_DurchH_end = seconds(); */
  /* double t_DurchH = t_DurchH_end - t_DurchH_start; */

  /* durchsatzHArr = nElem*1.e-9 /t_DurchH; */

  /* // verifiziren der Resultate */
  /* checkResult(hostRef, gpuRef, nElem); */

  // Device-Speicher freigeben
  CHECK(hipFree(d_A));
  CHECK(hipFree(d_B));
  CHECK(hipFree(d_C));

  // Host-Speicher freigeben
  free(h_A);
  free(h_B);
  free(hostRef);
  free(gpuRef);

  CHECK(hipDeviceReset());

  /* write results to file */
  fprintf(
          results_file,
          "%f, %f, %f, %f, %f\n",
          latenzHDArr, latenzDHArr,
          bandbreiteHDArr, bandbreiteDHArr,
          durchsatzDArr/* , durchsatzHArr */
          );
}

int main(int argc, char **argv)
{
  printf("%s Starting...\n", argv[0]);

  typedef struct grid_parameters {
    int gridX;
    int gridY;
    int threadX;
    int threadY;
  } grid_params_t;

  int data_points = 820;
  /* allocate 820 points for the grid data */
  grid_params_t grid_data[data_points];

  /* read in grid parameters */
  FILE *f = fopen("grid_parameters", "r");
  int i;
  for (i = 0;
       i != data_points &&
         fscanf(f, "%d, %d, %d, %d\n", &grid_data[i].gridX, &grid_data[i].gridY, &grid_data[i].threadX, &grid_data[i].threadY) != EOF;
       i++
       );
  fclose(f);

  // Device auswaehlen
  int dev = 0;
  CHECK(hipSetDevice(dev));

  /* // Groesse der arrays festlegen */
  int nElem = 1024*1024;

  /* overwrite results */
  FILE *g = fopen("grid_parameter_results", "w");

  fprintf(
          g,
          "%s, %s, %s, %s, %s\n",
          "latenzHDArr", "latenzDHArr",
          "bandbreiteHDArr", "bandbreiteDHArr",
          "durchsatzDArr"
          );

  int blockIdx;
  int blockIdy;
  int threadIdx;
  int threadIdy;

  for (i = 0; i < data_points+1; i++) {
    blockIdx = grid_data[i].gridX;
    blockIdy = grid_data[i].gridY;
    threadIdx = grid_data[i].threadX;
    threadIdy = grid_data[i].threadY;

    testGridParamter(nElem, g, blockIdx, blockIdy, threadIdx, threadIdy);
    printf("%d/%d\r", i, data_points);
    fflush(stdout);
  }
  printf("\n");

  /* close results */
  fclose(g);

  return(0);
}

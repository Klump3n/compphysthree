#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "spin.h"
#include "global.h"
#include "randgpu.h"

static spin b;

double action(void)
{
   int idx,k;
   double act,tmp,ka2;
   spin h2;

#ifdef DEBUG
   printf("lambda: %f, kappa: %f, h: %f + I %f\n",lambda,kappa,hipCreal(h),hipCimag(h));
#endif

   act=0.0;
   ka2=2.0*kappa;
   h2=make_spin(hipCreal(h)*2.0,hipCimag(h)*2.0);
   for (idx=0; idx<nvol; idx++)
   {
      tmp=hipCreal(phi[idx])*hipCreal(phi[idx]) + hipCimag(phi[idx])*hipCimag(phi[idx]);
      act+=tmp;
      tmp-=1.0;
      act+=lambda*tmp*tmp;
      tmp=0.0;
      for (k=1; k<=ndim; k++)
      {
         tmp+=(hipCreal(phi[idx])*hipCreal(phi[nn[k][idx]]) + hipCimag(phi[idx])*hipCimag(phi[nn[k][idx]]));
      }
      act-=ka2*tmp;
      act-=(hipCreal(phi[idx])*hipCreal(h2) + hipCimag(phi[idx])*hipCimag(h2));
   }

   return act;
}

void random_cnfg(void)
{
   int idx;
   double *rnd;

   rnd=randgpu(2*nvol);

   for (idx=0; idx<nvol; idx++)
   {
      phi[idx]=make_spin(rnd[2*idx]-0.5,(rnd[2*idx+1]-0.5));
   }
}

spin magnet(void)
{
   int idx;
   spin m;

   m=make_spin(0.0,0.0);
   for (idx=0; idx<nvol; idx++)
   {
      m=hipCadd(m,phi[idx]);
   }
   m=make_spin(hipCreal(m)/nvol,hipCimag(m)/nvol);

   return m;
}

void compute_b(int idx)
{
   int k;
   spin tmpc;

   b=h;
   for (k=1; k<=ndim; k++)
   {
      tmpc=hipCadd(phi[nn[k][idx]],phi[nn[ndim+k][idx]]);
      b=make_hipDoubleComplex(hipCreal(b)+kappa*hipCreal(tmpc),hipCimag(b)+kappa*hipCimag(tmpc));
   }
}

// use pre-computed b
double alocal2(int idx)
{
   double a,tmp;

   tmp=hipCreal(phi[idx])*hipCreal(phi[idx]) + hipCimag(phi[idx])*hipCimag(phi[idx]);
   a=2.0*(hipCreal(b)*hipCreal(phi[idx])+hipCimag(b)*hipCimag(phi[idx]))-tmp;
   tmp-=1.0;
   a-=lambda*tmp*tmp;

   #ifdef DEBUG
      printf("b: %f + %f i, a: %f\n",hipCreal(b),hipCimag(b),a);
   #endif

   return -a;
}

// compute b
double alocal(int idx)
{
   compute_b(idx);

   return alocal2(idx);
}

#include "hip/hip_runtime.h"
#define DEFINE_GLOBAL

#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include "global.h"
#include "randgpu.h"
#include "geom_pbc.h"
#include "metropolis.h"
#include "spin.h"
#include "common.h"
#include "added_stuff.h"
#include "added_stuff_gpu.h"

#include <hip/hip_runtime.h>

#define MIN_NARG 6

#define CHECK(call)                                             \
  {                                                             \
    const hipError_t error = call;                             \
    if (error != hipSuccess)                                   \
      {                                                         \
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);  \
        fprintf(stderr, "code: %d, reason: %s\n", error,        \
                hipGetErrorString(error));                     \
        exit(1);                                                \
      }                                                         \
  }

void usage(void)
{
   printf("Usage:\n\n");
   printf("  run <lambda> <kappa> <h> <phi0> <nsweep> <lsize1> [<lsize2> ...]\n\n");
   exit(0);
}

void init_phi(double phi0)
{
   int idx;

   if (phi0==0.0)
   {
      random_cnfg();
   }
   else
   {
      for (idx=0; idx<nvol; idx++)
      {
         phi[idx]=make_spin(phi0,0.0);
      }
   }
}

double tune_delta(double acc, double delta)
{
   if (acc<0.35)
      delta*=0.95;
   if (acc>0.45)
      delta*=1.05;

   return delta;
}

void gpu_stuff(int nsweep)
{

  // set up device
  int dev = 0;
  hipDeviceProp_t deviceProp;
  CHECK(hipGetDeviceProperties(&deviceProp, dev));
  printf("Using Device %d: %s\n", dev, deviceProp.name);
  CHECK(hipSetDevice(dev));

  int *d_nn;
  CHECK(hipMalloc((void**)&d_nn,nvol*(2*ndim+1)*sizeof(int)));
  CHECK(hipMemcpy(d_nn, nn[0], nvol*(2*ndim+1)*sizeof(int), hipMemcpyHostToDevice));

  CHECK(hipMemcpyToSymbol(HIP_SYMBOL(devLambda), &lambda, sizeof(double)));
  CHECK(hipMemcpyToSymbol(HIP_SYMBOL(devKappa), &kappa, sizeof(double)));
  CHECK(hipMemcpyToSymbol(HIP_SYMBOL(devNdim), &ndim, sizeof(int)));
  CHECK(hipMemcpyToSymbol(HIP_SYMBOL(devNvol), &nvol, sizeof(int)));

  int halfArrayLength = ceil((float) (nvol) / 2.);

  int *evenArray = (int *) calloc(halfArrayLength, sizeof(int));
  int *oddArray = (int *) calloc(halfArrayLength, sizeof(int));
  evenOddIndices(evenArray, oddArray);

  int *d_evenArray;
  CHECK(hipMalloc((void**)&d_evenArray, halfArrayLength * sizeof(int)));
  CHECK(hipMemcpy(d_evenArray, evenArray, halfArrayLength * sizeof(int), hipMemcpyHostToDevice));
  int *d_oddArray;
  CHECK(hipMalloc((void**)&d_oddArray, halfArrayLength * sizeof(int)));
  CHECK(hipMemcpy(d_oddArray, oddArray, halfArrayLength * sizeof(int), hipMemcpyHostToDevice));

  spin *d_bEvenArray;
  CHECK(hipMalloc((void**)&d_bEvenArray, halfArrayLength * sizeof(spin)));
  CHECK(hipMemset(d_bEvenArray, 0, halfArrayLength * sizeof(spin)));
  spin *d_bOddArray;
  CHECK(hipMalloc((void**)&d_bOddArray, halfArrayLength * sizeof(spin)));
  CHECK(hipMemset(d_bOddArray, 0, halfArrayLength * sizeof(spin)));

  spin *d_phi;
  CHECK(hipMalloc((void**)&d_phi, nvol*sizeof(spin)));
  CHECK(hipMemcpy(d_phi, phi, nvol*sizeof(spin), hipMemcpyHostToDevice));

  int *d_accept;
  CHECK(hipMalloc((void**)&d_accept, halfArrayLength * sizeof(int)));
  CHECK(hipMemset(d_accept, 0, halfArrayLength * sizeof(int)));

  spin *d_phi_intermediate;
  CHECK(hipMalloc((void**)&d_phi_intermediate, nvol*sizeof(spin)));
  CHECK(hipMemset(d_phi_intermediate, 0, nvol*sizeof(spin)));

  double *d_aloc_comp;
  CHECK(hipMalloc((void**)&d_aloc_comp, halfArrayLength * sizeof(double)));
  CHECK(hipMemset(d_aloc_comp, 0, halfArrayLength * sizeof(double)));

  double *d_aloc_calc;
  CHECK(hipMalloc((void**)&d_aloc_calc, halfArrayLength * sizeof(double)));
  CHECK(hipMemset(d_aloc_calc, 0, halfArrayLength * sizeof(double)));

  /* NTRIAL IS 10!!! */
  double *d_rnd = randgpu_device_ptr(3*nvol*10);

  double *rnd = (double *) calloc(nvol, sizeof(double));
  CHECK(hipMemcpy(rnd, d_rnd, nvol*3*10*sizeof(double), hipMemcpyDeviceToHost));
  /* for (int i=0; i<nvol*30; i++) */
  /*   { */
  /*     printf("%d, %f\n", i, doublearray[i]); */
  /*   } */

  spin m, gpu_m;
  double mm, gpu_mm;

  double delta = .2;
  double cpu_delta = .2;

  double acc = 0.0;
  double cpu_acc = 0.0;

  spin *backup_phi = (spin *) malloc(nvol*sizeof(spin));

  printf("ITER \t ACC \t\t ACC_C \t\t DELTA \t\t DELTA_C \t GPU_MM \t CPU_MM\n");
  for (int i=1; i<=nsweep; i++)
    {

//      cpu_acc = metro_sweep_alt(cpu_delta, evenArray, oddArray, rnd);
//      cpu_delta=tune_delta(cpu_acc,cpu_delta);
//
//      m=magnet();
//      mm=hipCabs(m)*hipCabs(m);

      acc=gpu_sweep(d_phi,
                    d_evenArray,
                    d_oddArray,
                    d_bEvenArray,
                    d_bOddArray,
                    d_nn,
                    d_accept,
                    d_phi_intermediate,
                    d_aloc_comp,
                    d_aloc_calc,
                    d_rnd,
                    delta
                    );

//      memcpy(backup_phi, phi, nvol*sizeof(spin)); /* keep original phi */
//      CHECK(hipMemcpy(phi, d_phi, nvol*sizeof(spin), hipMemcpyDeviceToHost));
      gpu_m = magnet();
//      memcpy(phi, backup_phi, nvol*sizeof(spin)); /* restore phi */

      gpu_mm=hipCabs(gpu_m)*hipCabs(gpu_m);

      delta=tune_delta(acc,delta);

      /* printf("%d\t %f\t %f\t %f\t %f\t %f\t %f\n",i,acc,delta,s,hipCreal(m),hipCimag(m),mm); */

      printf("%d \t %f \t %f \t %f \t %f \t %f \t %f \n", i, acc, cpu_acc, delta, cpu_delta, gpu_mm, mm);
    }

}

int main(int argc, char **argv)
{
   printf("%s Starting...\n", argv[0]);

   int i, nsweep;
   double acc, delta, s, iStart, mm, phi0, reh;
   spin m;

   if (argc<MIN_NARG+1)
      usage();

   // read parameters from command line
   lambda=atof(argv[1]);
   kappa=atof(argv[2]);
   reh=atof(argv[3]);
   phi0=atof(argv[4]);
   nsweep=atoi(argv[5]);

   ndim=argc-MIN_NARG;
   lsize=(int*)malloc((ndim+1)*sizeof(int));
   for (i=1; i<=ndim; i++)
   {
      lsize[i]=atoi(argv[i+MIN_NARG-1]);
   }

   delta=0.2;
   h=make_spin(reh,0.0);
   CHECK(hipMemcpyToSymbol(HIP_SYMBOL(devH), &h, sizeof(spin)));

   // print out parameters
   printf("Gittergroesse: %d",lsize[1]);
   for (i=2; i<=ndim; i++)
   {
      printf(" x %d",lsize[i]);
   }
   printf("\n\n");
   printf("nsweep = %d\n",nsweep);
   printf("lambda = %f\n",lambda);
   printf("kappa  = %f\n",kappa);
   printf("delta  = %f\n",delta);
   printf("h      = %f + I %f\n",hipCreal(h),hipCimag(h));
   if (phi0==0.0)
      printf("phi    = random\n");
   else
      printf("phi    = %f + I %f\n",phi0,0.0);
   printf("\n\n");

   // set up geometry
   geom_pbc();

   // allocate spins, set random values
   printf("Initalize spins... ");
   iStart=seconds();
   phi=(spin*)malloc(nvol*sizeof(spin));
   init_phi(phi0);
   printf("%f sec.\n\n",seconds()-iStart);


   spin *initial_phi = (spin *) malloc(nvol*sizeof(spin));
   memcpy(initial_phi, phi, nvol*sizeof(spin)); /* keep original phi */
   /* printf("i phi init_phi\n"); */
   /* for (int i=0; i<nvol; i++) */
   /*   { */
   /*     printf("%d, %f, %f\n", i, hipCabs(phi[i]), hipCabs(initial_phi[i])); */
   /*   } */


   /* s=action(); */
   /* m=magnet(); */
   /* mm=hipCabs(m)*hipCabs(m); */

   /* printf("UPD\t A       \t DELTA   \t S       \t RE(M)   \t IM(M)   \t |M|^2   \n"); */
   /* printf("%d\t %f\t %f\t %f\t %f\t %f\t %f\n",0,0.0,delta,s,hipCreal(m),hipCimag(m),mm); */

   /* acc=0.0; */
   /* iStart=seconds(); */
   /* for (i=1; i<=nsweep; i++) */
   /* { */
   /*    acc=metro_sweep(delta); */
   /*    delta=tune_delta(acc,delta); */
   /*    s=action(); */
   /*    m=magnet(); */
   /*    mm=hipCabs(m)*hipCabs(m); */
   /*    printf("%d\t %f\t %f\t %f\t %f\t %f\t %f\n",i,acc,delta,s,hipCreal(m),hipCimag(m),mm); */
   /* } */

   /* printf("\n\n"); */
   /* printf("%d updates took %f sec.\n\n",nsweep,seconds()-iStart); */



   memcpy(phi, initial_phi, nvol*sizeof(spin)); /* restore phi */
   /* printf("i phi init_phi\n"); */
   /* for (int i=0; i<nvol; i++) */
   /*   { */
   /*     printf("%d, %f, %f\n", i, hipCabs(phi[i]), hipCabs(initial_phi[i])); */
   /*   } */
   gpu_stuff(nsweep);





   free(lsize);
   free(nn[0]);
   free(nn);
   free(phi);
}

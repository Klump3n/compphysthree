#include "hip/hip_runtime.h"
#include "spin.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "global.h"
#include "randgpu.h"
#include "added_stuff.h"
#include "added_stuff_gpu.h"

#define CHECK(call)                                             \
  {                                                             \
    const hipError_t error = call;                             \
    if (error != hipSuccess)                                   \
      {                                                         \
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);  \
        fprintf(stderr, "code: %d, reason: %s\n", error,        \
                hipGetErrorString(error));                     \
        exit(1);                                                \
      }                                                         \
  }

double gpu_sweep(
                 spin *d_phi,
                 int *d_evenArray,
                 int *d_oddArray,
                 spin *d_bEvenArray,
                 spin *d_bOddArray,
                 int *d_nn,
                 int *d_accept,
                 spin *d_phi_intermediate,
                 double *d_aloc_comp,
                 double *d_aloc_calc,
                 double *d_rnd,
                 double delta
                 )
{

  delta = 2.0 * delta;          /* consistency */

  int ntrial = 10;
  int acc_sum = 0;
  dim3 blockSize;

  int halfArrayLength = ceil((float) (nvol) / 2.);

  int *h_accept = (int *) calloc(halfArrayLength, sizeof(int));

  if (nvol/2 < 128)
    {
      blockSize.x = halfArrayLength;
    }
  else
    {
      blockSize.x = 128;
    }
  blockSize.y = 1;
  blockSize.z = 1;

  dim3 gridSize;
  gridSize.x = ceil((float) nvol / (float) 256);
  gridSize.y = 1;
  gridSize.z = 1;

  /* reset values of helper arrays */
  CHECK(hipMemset(d_accept, 0, halfArrayLength*sizeof(int)));
  CHECK(hipMemset(d_phi_intermediate, 0, nvol*sizeof(spin)));
  CHECK(hipMemset(d_aloc_comp, 0.0, halfArrayLength*sizeof(double)));
  CHECK(hipMemset(d_aloc_calc, 0.0, halfArrayLength*sizeof(double)));

  /* even sweep */
  /* b even */
  /**/
  compute_b_gpu<<<blockSize, gridSize>>>(d_phi, d_nn, d_bEvenArray);
  CHECK(hipDeviceSynchronize());

  alocal_gpu<<<gridSize, blockSize>>>(d_phi, d_bEvenArray, d_aloc_comp);
  CHECK(hipDeviceSynchronize());

  for (int i=0; i<ntrial; i++)
    {
      CHECK(hipMemcpy(d_phi_intermediate, d_phi, nvol*sizeof(spin), hipMemcpyDeviceToDevice));
      modify_spin<<<gridSize, blockSize>>>(d_phi, d_rnd, delta, i);
      CHECK(hipDeviceSynchronize());
      alocal_gpu<<<gridSize, blockSize>>>(d_phi, d_bEvenArray, d_aloc_calc);
      CHECK(hipDeviceSynchronize());
      gpu_comp_action <<<gridSize, blockSize>>> (d_phi, d_phi_intermediate, d_aloc_comp, d_aloc_calc, d_rnd, i, d_accept);
      CHECK(hipDeviceSynchronize());
    }

  /* get d_accept and sum it all up */
  CHECK(hipMemcpy(h_accept, d_accept, ((int)(nvol/2))*sizeof(int), hipMemcpyDeviceToHost));
  for (int i=0; i<((int)(nvol/2)); i++)
    {
      acc_sum += h_accept[i];
    }

  /* end even */

  /* reset values of helper arrays */
  CHECK(hipMemset(d_accept, 0, halfArrayLength*sizeof(int)));
  CHECK(hipMemset(d_phi_intermediate, 0, nvol*sizeof(spin)));
  CHECK(hipMemset(d_aloc_comp, 0.0, halfArrayLength*sizeof(double)));
  CHECK(hipMemset(d_aloc_calc, 0.0, halfArrayLength*sizeof(double)));

  /* odd sweep */
  /* b odd */
  /**/
  compute_b_gpu<<<blockSize, gridSize>>>(d_phi, d_nn, d_bOddArray);
  CHECK(hipDeviceSynchronize());

  alocal_gpu<<<gridSize, blockSize>>>(d_phi, d_bOddArray, d_aloc_comp);
  CHECK(hipDeviceSynchronize());

  for (int i=0; i<ntrial; i++)
    {
      /* backup_spin <<<gridSize, blockSize>>> (d_phi, d_phi_intermediate); */
      /* CHECK(hipDeviceSynchronize()); */
      CHECK(hipMemcpy(d_phi_intermediate, d_phi, nvol*sizeof(spin), hipMemcpyDeviceToDevice)); /* faster */
      modify_spin<<<gridSize, blockSize>>>(d_phi, d_rnd, delta, i);
      CHECK(hipDeviceSynchronize());
      alocal_gpu<<<gridSize, blockSize>>>(d_phi, d_bOddArray, d_aloc_calc);
      CHECK(hipDeviceSynchronize());
      gpu_comp_action <<<gridSize, blockSize>>> (d_phi, d_phi_intermediate, d_aloc_comp, d_aloc_calc, d_rnd, i, d_accept);
      CHECK(hipDeviceSynchronize());
    }

  /* get d_accept and sum it all up */
  CHECK(hipMemcpy(h_accept, d_accept, halfArrayLength*sizeof(int), hipMemcpyDeviceToHost));
  for (int i=0; i<((int)(nvol/2)); i++)
    {
      acc_sum += h_accept[i];
    }

  /* end even */
  double percentage = (float) (acc_sum)/ (float) (ntrial*nvol);
  /* printf("%d of %d: %f\n", acc_sum, ntrial*nvol, percentage); */
  return percentage;

}

__global__
void backup_spin(spin *d_phi, spin *d_phi_intermediate)
{
  unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  d_phi_intermediate[idx] = d_phi[idx];
}

__global__
void compute_b_gpu(spin *d_phi, int *d_nn, spin *d_bArray)
{

  unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

  spin tmpc;

  d_bArray[idx] = devH;
  for (int k=1; k<=devNdim; k++)
    {
      tmpc = hipCadd(d_phi[d_nn[k*devNvol+ idx]], d_phi[d_nn[(devNdim+k)*devNvol + idx]]);
      d_bArray[idx] = make_hipDoubleComplex(hipCreal(d_bArray[idx])+devKappa*hipCreal(tmpc), hipCimag(d_bArray[idx])+devKappa*hipCimag(tmpc));
    }
}

__global__
void gpu_comp_action(spin *d_phi, spin *d_phi_intermediate, double *d_aloc_comp, double *d_aloc_calc, double *d_rnd, int ktrial, int *d_accept)
{
  unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

  /* NTRIAL = 10!!! */
  int rnd_idx = ktrial*10 + idx + 2;

  if (d_rnd[rnd_idx] < exp(-d_aloc_calc[idx] + d_aloc_comp[idx]))
    {
      d_accept[idx]++;
      d_aloc_comp[idx] = d_aloc_calc[idx];
    }
  else
    {
      d_phi[idx] = d_phi_intermediate[idx];
    }

}

__global__
void modify_spin(spin *d_phi, double *d_rnd, double delta, int ktrial)
{
  unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

  /* NTRIAL = 10!!! */
  int rnd_idx = ktrial*10 + idx;

  d_phi[idx]=make_spin(
                       hipCreal(d_phi[idx])+delta*(d_rnd[rnd_idx + 0]-0.5),
                       hipCimag(d_phi[idx])+delta*(d_rnd[rnd_idx + 1]-0.5)
                       );

}

/* __global__ */
/* spin magnet(void) */
/* { */
/*   int idx; */
/*   spin m; */

/*   m=make_spin(0.0,0.0); */
/*   for (idx=0; idx<nvol; idx++) */
/*     { */
/*       m=hipCadd(m,phi[idx]); */
/*     } */
/*   m=make_spin(hipCreal(m)/nvol,hipCimag(m)/nvol); */

/*   return m; */
/* } */

// use pre-computed b
__global__
void alocal_gpu(spin *d_phi, spin *d_bArray, double *d_aloc)
{
  unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

  double a,tmp;

  tmp=hipCreal(d_phi[idx])*hipCreal(d_phi[idx]) + hipCimag(d_phi[idx])*hipCimag(d_phi[idx]);
  a=2.0*(hipCreal(d_bArray[idx])*hipCreal(d_phi[idx])+hipCimag(d_bArray[idx])*hipCimag(d_phi[idx]))-tmp;
  tmp-=1.0;
  a-=devLambda*tmp*tmp;

  d_aloc[idx] = -a;
}

#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>

/*
 * Dieses Beispiel demonstriert die Addition zweier Arrays.
 * addArrayGPU soll die Arbeit ueber CUDA Threads auf der GPU verteilen.
 * addArrayHost iteriert sequentiell durch die Vektorelemente auf dem Host.
 */

// Macro zur Fehlerauswertung
#define CHECK(call)                                                            \
{                                                                              \
    const hipError_t error = call;                                            \
    if (error != hipSuccess)                                                  \
    {                                                                          \
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                 \
        fprintf(stderr, "code: %d, reason: %s\n", error,                       \
                hipGetErrorString(error));                                    \
        exit(1);                                                               \
    }                                                                          \
}

double seconds()
{
    struct timeval tp;
    struct timezone tzp;
    int i = gettimeofday(&tp, &tzp);
    return ((double)tp.tv_sec + (double)tp.tv_usec * 1.e-6);
}

void checkResult(float *hostRef, float *gpuRef, const int N)
{
    double epsilon = 1.0E-8;
    bool match = 1;

    for (int i = 0; i < N; i++)
    {
        if (abs(hostRef[i] - gpuRef[i]) > epsilon)
        {
            match = 0;
            printf("Arrays stimmen nicht ueberein!\n");
            printf("host %5.2f gpu %5.2f an der Stelle %d\n", hostRef[i],
                   gpuRef[i], i);
            break;
        }
    }

    if (match) printf("Arrays stimmen ueberein.\n\n");

    return;
}

void initialData(float *ip, int size)
{
    // erzeuge zufaellige Eintraege
    time_t t;
    srand((unsigned) time(&t));

    for (int i = 0; i < size; i++)
    {
        ip[i] = (float)(rand() & 0xFF) / 10.0f;
    }

    return;
}


void addArrayHost(float *A, float *B, float *C, const int N)
{
    for (int idx = 0; idx < N; idx++)
        C[idx] = A[idx] + B[idx];
}

__global__ void addArrayGPU(float *A, float *B, float *C)
{
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  C[idx] = A[idx] + B[idx];
}


int main(int argc, char **argv)
{
    printf("%s Starting...\n", argv[0]);

    // Device auswaehlen
    int dev = 0;
    CHECK(hipSetDevice(dev));

    // Groesse der arrays festlegen
    int nElem = 1024;
    if (argc>1)
      nElem = atoi(argv[1]);
    printf("Array-Groesse: %d\n", nElem);

    // Host-Speicher allozieren mit malloc
    size_t nBytes = nElem * sizeof(float);

    float *h_A, *h_B, *hostRef, *gpuRef;
    h_A     = (float *)malloc(nBytes);
    h_B     = (float *)malloc(nBytes);
    hostRef = (float *)malloc(nBytes);
    gpuRef  = (float *)malloc(nBytes);

    // initialisiere Arrays auf dem Host
    initialData(h_A, nElem);
    initialData(h_B, nElem);

    memset(hostRef, 0, nBytes);
    memset(gpuRef,  0, nBytes);

    // Device-Speicher allozieren mit hipMalloc
    float *d_A, *d_B, *d_C;
    CHECK(hipMalloc((float**)&d_A, nBytes));
    CHECK(hipMalloc((float**)&d_B, nBytes));
    CHECK(hipMalloc((float**)&d_C, nBytes));

    // Starte Zeitmessung Latenz Host->Device
    double t_HDstart = seconds();
    
    // kopieren Host -> Device mit hipMemcpy
    CHECK(hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_C, gpuRef, nBytes, hipMemcpyHostToDevice));
    
    // Beende Zeitmessung Latenz Host->Device
    double t_HDend = seconds();
    
    double t_HD = (t_HDend-t_HDstart)*1.e+3;
    printf("Kopieren Host -> Device: %f ms\n", t_HD);

    // Berechne Bandbreite aus Latenz und Groesse der Arrays
    float bandbreite = 3*nElem*sizeof(float)/t_HD*1.e-9*1.e+3; // GByte/s
    //printf(" Groesse float %lu \n", sizeof(float));
    printf("Bandbreite: %f GB/s\n", bandbreite);
    
    //Starte Zeitmessung Durchsatz Device
    double t_DurchD_start = seconds();
    
    /* blockSize * threadSize HAS to be larger than nElem */
    /* int blockSize = 3; */
    int blockSize = (int) (nElem / 1024) + 1;
    printf("Blocksize %d\n", blockSize);
    int threadSize = 1024;
    addArrayGPU<<<blockSize, threadSize>>>(d_A, d_B, d_C);
    hipDeviceSynchronize();
    
    //Beende Zeitmessung Durchsatz Device
    double t_DurchD_end = seconds();
    double t_DurchD = t_DurchD_end - t_DurchD_start;
    
    double durchsatz = nElem*1.e-9 /t_DurchD;
    printf("Durchsatz in %f Gflops \n", durchsatz);

    // Starte Zeitmessung Latenz Device->Host
    double t_DHstart = seconds();
    
    // kopieren Device -> Host mit hipMemcpy
    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));
    
    // Beende Zeitmessung Latenz Device->Host
    double t_DHend = seconds();
    
    double t_DH = (t_DHend-t_DHstart)*1.e+3;
    printf("Kopieren Device -> Host: %f ms\n", t_DH);

    // Addition auf dem Host
    addArrayHost(h_A, h_B, hostRef, nElem);

    // verifiziren der Resultate
    checkResult(hostRef, gpuRef, nElem);

    // Device-Speicher freigeben
    CHECK(hipFree(d_A));
    CHECK(hipFree(d_B));
    CHECK(hipFree(d_C));

    // Host-Speicher freigeben
    free(h_A);
    free(h_B);
    free(hostRef);
    free(gpuRef);

    CHECK(hipDeviceReset());
    return(0);
}

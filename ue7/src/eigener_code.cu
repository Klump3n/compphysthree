#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "global.h"

#include "randgpu.h"
#include "action.h"


hipDoubleComplex mag(hipDoubleComplex *phi) {
  int idx;
  hipDoubleComplex magRes = make_hipDoubleComplex(0.0, 0.0);

  for (idx=0; idx<nvol; idx++){
    magRes = hipCadd(magRes, phi[idx]);
  }

  return hipCdiv(magRes, make_hipDoubleComplex((double)nvol, 0.0));
}

hipDoubleComplex spin_update(hipDoubleComplex *phi, double delta) {
  int idx;

  double *random_nums;
  random_nums = (double *) malloc(2*nvol*sizeof(double));
  random_nums = randgpu(2*nvol);

  for (idx=0; idx<nvol; idx++) {
    /* phi[idx] = phi[idx] + make_hipDoubleComplex( */
    /*                                            delta * (2 * random_nums[2*idx] + 1), */
    /*                                            delta * (2 * random_nums[2*idx + 1] + 1) */
    /*                                            ); */
  }

  return *phi;
}

/* hipDoubleComplex sweep() { */
  
/* } */

hipDoubleComplex spin_update_one_point(int idx, hipDoubleComplex *phi, double delta, double rand1, double rand2) {
  return hipCadd(phi[idx],
                make_hipDoubleComplex(
                                     delta * (2 * rand1 + 1),
                                     delta * (2 * rand2 + 1)
                                     )
                );
}

double akzeptanz(int idx, double delta, double rand1, double rand2) {

  hipDoubleComplex old_phi = phi[idx];

  double lambda = 1.0;
  double kappa = 1.0;
  hipDoubleComplex h = make_hipDoubleComplex(1.0, 1.0);

  double aloc_old_phi = alocal(idx, lambda, kappa, h);
  printf("aloc_old_phi = %f\n", aloc_old_phi);

  phi[idx] = spin_update_one_point(idx, phi, delta, rand1, rand2);

  double aloc_new_phi = alocal(idx, lambda, kappa, h);
  printf("aloc_new_phi = %f\n", aloc_new_phi);

  if (aloc_old_phi <= aloc_new_phi) {
    return 1.0;
  } else {
    phi[idx] = old_phi;
    return exp(aloc_new_phi - aloc_old_phi);
  }
}

#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_complex.h>
#include "global.h"

double S(hipDoubleComplex *phi, hipDoubleComplex h, double kappa, double lambda) {

  /*
    Folgende Groessen muessen gesetzt sein:             B Bunk 12/2005
    Dimension     ndim                              rev     4/2013
    Gittergroesse lsize[k], k=1..ndim

    Angelegt und berechnet wird
    Volumen       nvol
    NN-Indexfeld  nn[k][i], k=0..2*ndim, i=0..(nvol-1)

    nn[k][i] gibt den Index des Nachbarn von i in Richtung +k,
    unter Beruecksichtigung periodischer Randbedingungen.
    Fuer einen Schritt in Richtung -k setze man den Index auf (ndim+k).
    nn[0][i] ist reserviert.
  */
  double S_val = 0.0;

  double kappa_sum = 0.0;
  double phi_norm;

  /* nvol aus global */
  for (int i=0; i<nvol; i++)
    {

      kappa_sum = 0.0;
      /* mmmmmh spaghetticode */
      for (int j=0; j<ndim; j++) {
        kappa_sum += 2 * (hipCreal(phi[i]) * hipCreal(phi[ nn[j][i] ]) +
                          hipCimag(phi[i]) * hipCimag(phi[ nn[j][i] ]));
      }

      phi_norm = hipCabs(phi[i])*hipCabs(phi[i]);
      
      S_val +=
        phi_norm +
        lambda * (phi_norm - 1) * (phi_norm - 1) -
        kappa * kappa_sum -
        2 * (hipCreal(h) * hipCreal(phi[i]) +
             hipCimag(h) * hipCimag(phi[i]));
        }

  return S_val;
}

double S_analytical(hipDoubleComplex z, hipDoubleComplex h, double kappa, double lambda) {
  return nvol * (
                 (1 - 2*kappa*ndim) * hipCabs(z)*hipCabs(z) +
                 lambda * (hipCabs(z)*hipCabs(z) - 1) * (hipCabs(z)*hipCabs(z) - 1) -
                 2 * (hipCreal(h) * hipCreal(z) +
                      hipCimag(h) * hipCimag(z))
                 );
}

double p(int x, hipDoubleComplex phi_x, hipDoubleComplex h, double kappa, double lambda) {

  /* hipDoubleComplex phi_x = phi[x]; */
  double phi_norm = hipCabs(phi_x)*hipCabs(phi_x);
  hipDoubleComplex kappa_sum = make_hipDoubleComplex(0.0, 0.0);

  for (int j=0; j<ndim; j++) {
    kappa_sum = hipCadd(kappa_sum,
                       hipCadd(phi[ nn[j][x] ], phi[ nn[ndim+j][x] ])
                       );
  };
  hipDoubleComplex Bx = hipCadd(h, kappa_sum);

  return
    /* exp( */
        2 * (hipCreal(Bx) * hipCreal(phi_x) + hipCimag(Bx) * hipCimag(phi_x)) -
        phi_norm - lambda * (phi_norm - 1) * (phi_norm - 1)
        /* ) */;

}

/* double p_comp(hipDoubleComplex phi_one, hipDoubleComplex phi_two, int index, double lambda) { */
/*   double phi_norm_one = hipCabs(phi_one)*hipCabs(phi_one); */
/*   double phi_norm_two = hipCabs(phi_two)*hipCabs(phi_two); */

/*   hipDoubleComplex kappa_sum = make_hipDoubleComplex(0.0, 0.0); */
/*   for (int j=0; j<ndim; j++) { */
/*     kappa_sum = hipCadd(kappa_sum, */
/*                        hipCadd(phi_one[ nn[j][index] ], phi_one[ nn[ndim+j][index] ]) */
/*                        ); */
/*   }; */
/*   hipDoubleComplex Bx = hipCadd(h, kappa_sum); */
/*   double asdf = 2 * (hipCreal()); */
/* } */

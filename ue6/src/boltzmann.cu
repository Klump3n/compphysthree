#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_complex.h>
#include "global.h"

double S(hipDoubleComplex *phi, hipDoubleComplex h, double kappa, double lambda) {

  /*
    Folgende Groessen muessen gesetzt sein:             B Bunk 12/2005
    Dimension     ndim                              rev     4/2013
    Gittergroesse lsize[k], k=1..ndim

    Angelegt und berechnet wird
    Volumen       nvol
    NN-Indexfeld  nn[k][i], k=0..2*ndim, i=0..(nvol-1)

    nn[k][i] gibt den Index des Nachbarn von i in Richtung +k,
    unter Beruecksichtigung periodischer Randbedingungen.
    Fuer einen Schritt in Richtung -k setze man den Index auf (ndim+k).
    nn[0][i] ist reserviert.
  */
  double S_val = 0.0;

  double kappa_sum = 0.0;
  double phi_norm;

  /* nvol aus global */
  for (int i=0; i<nvol; i++)
    {

      kappa_sum = 0.0;

      for (int j=1; j<(ndim+1); j++) {
        kappa_sum += 2 * (hipCreal(phi[i]) * hipCreal(phi[ nn[j][i] ]) +
                          hipCimag(phi[i]) * hipCimag(phi[ nn[j][i] ]));
      }

      phi_norm = hipCabs(phi[i])*hipCabs(phi[i]);

      S_val +=
        phi_norm +
        lambda * (phi_norm - 1) * (phi_norm - 1) -
        kappa * kappa_sum -
        2 * (hipCreal(h) * hipCreal(phi[i]) +
             hipCimag(h) * hipCimag(phi[i]));
        }

  return S_val;
}

double S_analytical(hipDoubleComplex z, hipDoubleComplex h, double kappa, double lambda) {
  return nvol * (
                 (1 - 2*kappa*ndim) * hipCabs(z)*hipCabs(z) +
                 lambda * (hipCabs(z)*hipCabs(z) - 1) * (hipCabs(z)*hipCabs(z) - 1) -
                 2 * (hipCreal(h) * hipCreal(z) +
                      hipCimag(h) * hipCimag(z))
                 );
}

double p_arg(int x, hipDoubleComplex *phi, hipDoubleComplex h, double kappa, double lambda) {

  hipDoubleComplex phi_x = phi[x];

  double phi_norm = hipCabs(phi_x)*hipCabs(phi_x);
  hipDoubleComplex kappa_sum = make_hipDoubleComplex(0.0, 0.0);

  for (int j=1; j<(ndim + 1); j++) {
    kappa_sum = hipCadd(kappa_sum,
                       hipCadd(phi[ nn[j][x] ], phi[ nn[ndim+j][x] ])
                       );
  };
  kappa_sum = hipCmul(make_hipDoubleComplex(kappa, 0.0), kappa_sum);
  hipDoubleComplex Bx = hipCadd(h, kappa_sum);

  return 2 * (hipCreal(Bx) * hipCreal(phi_x) + hipCimag(Bx) * hipCimag(phi_x)) -
    phi_norm - lambda * (phi_norm - 1) * (phi_norm - 1);
}

double p(int x, hipDoubleComplex *phi, hipDoubleComplex h, double kappa, double lambda) {
  return exp(p_arg(x, phi, h, kappa, lambda));
}

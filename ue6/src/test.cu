
#define DEFINE_GLOBAL

#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_complex.h>
#include "global.h"

int main(void)
{
   int k;
   double r;
   hipDoubleComplex z = make_hipDoubleComplex(1.5,2.0);

   printf("  z    = %f + %f * i\n", hipCreal(z), hipCimag(z));
   r=hipCabs(z);
   printf(" |z|^2 = %.6f\n", r*r);
   z=hipCmul(z,hipConj(z));
   printf("  zz^* = %.6f + %.6f\n",  hipCreal(z), hipCimag(z));

   phi=(hipDoubleComplex*)malloc(10*sizeof(hipDoubleComplex));

   for (k=0; k<10; k++)
   {
      phi[k]=make_hipDoubleComplex((double)k,(double)(k*k));
      printf("%f + %f * i\n", hipCreal(phi[k]), hipCimag(phi[k]));
   }
}
